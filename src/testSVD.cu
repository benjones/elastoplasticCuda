#include "hip/hip_runtime.h"

#include "matLib.h"

void __global__ svdKernel(const mat3& A, mat3* U, vec3* S, mat3* V){


  SVD(A, *U, *S, *V);

  

}


int main(int argc, char ** argv){


  mat3 A_h = {1,0,0, 0, 1, 0, 0, 0, 1};

  mat3 U_h, V_h;
  vec3 S_h;

  mat3 *A_d, *U_d, *V_d;
  vec3 *S_d;

  hipMalloc(&A_d, sizeof(mat3));
  hipMalloc(&U_d, sizeof(mat3));
  hipMalloc(&V_d, sizeof(mat3));
  hipMalloc(&S_d, sizeof(vec3));
  
  hipMemcpy(A_d, &A_h, sizeof(mat3), hipMemcpyHostToDevice);

  svdKernel<<<1,1>>>(A_d, U_d, S_d, V_d);

  hipMemcpy(&U_h, U_d, sizeof(mat3), hipMemcpyDeviceToHost);
  hipMemcpy(&V_h, V_d, sizeof(mat3), hipMemcpyDeviceToHost);
  hipMemcpy(&S_h, S_d, sizeof(vec3), hipMemcpyDeviceToHost);

  printVector(S_h);

  return 0;
}
