#include "hip/hip_runtime.h"

#include "world.cuh"


__global__ void step(vec3* positions,
	   	   vec3* velocities,
	   	   float dt){


  positions[threadIdx.x].x += velocities[threadIdx.x].x*dt;
  positions[threadIdx.x].y += velocities[threadIdx.x].y*dt;
  positions[threadIdx.x].z += velocities[threadIdx.x].z*dt;

}

// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel(
		float4 *dptr,
		int numParticles, 
		vec3* positions,
	   	vec3* velocities,
	   	float dt)
{
    // execute the kernel
   	step<<<numParticles, 1>>>(positions, velocities, dt);
}
