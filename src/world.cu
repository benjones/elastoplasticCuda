#include "hip/hip_runtime.h"

#include "world.cuh"
#include "matLib.h"

const float kernelRadius = .6f;

const float density = 1000.0f;

const float lambda = 10000.0f;
const float mu = 10000.0f;


__global__ void step(
			int numParticles,
			float4* positions,
	   	   	float4* velocities,
			float4* embedded,
	   	   	float dt){


	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;

	if (idx >= numParticles) return;	// out of bounds


	

	//compute basis matrix;
	mat3 A = matZero();

	mat3 rhs1 = matZero();
	mat3 rhs2 = matZero();
	

	float wSum = 0;
	for(int i = 0; i < numParticles; ++i){
	  if( i != idx){
	    float4 vij = vecSub(embedded[i], embedded[idx]) ;
	    float wij = sphKernel(kernelRadius, vecMag(vij));
	    A = matAdd(A, matScale(outerProduct(vij, vij), wij));
	    
	    rhs1 = matAdd(rhs1, matScale(outerProduct(vecSub(positions[i], positions[idx]), vij), wij));
	    rhs1 = matAdd(rhs1, matScale(outerProduct(vecSub(velocities[i], velocities[idx]), vij), wij));
	    
	    
	    wSum += wij;
	    
	  }
	}
	
	mat3 AU, AV;
	float4 AS;
	SVD(A, AU, AS, AV);


	float volume = sqrtf(AS.x*AS.y*AS.z/(1 + wSum*wSum*wSum));
	float mass = density/volume;

	mat3 Ainv = pseudoInverse(AU, AS, AV);;
	mat3 F = matMult(rhs1,Ainv);
	mat3 FDot = matMult(rhs2,Ainv);

	mat3 FU, FV;
	float4 FS;
	SVD(F, FU, FS, FV);
	float4 ones = make_float4(1.0f, 1.0f, 1.0f, 0.0f);
	float4 strain = vecSub(FS, ones);
	float lTrace = lambda*(strain.x + strain.y + strain.z);
	mat3 stress = matScale(matDiag(strain), 2*mu);
	stress.m00 += lTrace;
	stress.m11 += lTrace;
	stress.m22 += lTrace;


	//un-diagonalize
	stress = matMult(matMult(FU, stress), matTranspose(FV));
	

	


  	positions[idx].x += velocities[idx].x*dt;
  	positions[idx].y += velocities[idx].y*dt;
  	positions[idx].z += velocities[idx].z*dt;



	// check boundaries
	if(positions[idx].y < GROUND_HEIGHT) positions[idx].y = GROUND_HEIGHT;
}

// Wrapper for the __global__ call that sets up the kernel call
extern "C" void launch_kernel(
		int numParticles, 
		float4* positions,
	   	float4* velocities,
		float4* embedded,
	   	float dt)
{
	dim3 threadLayout(BLOCK_SIZE, 1, 1);
	int blockCnt = numParticles / BLOCK_SIZE;
	if(blockCnt*BLOCK_SIZE < numParticles) blockCnt++;
	dim3 blockLayout(blockCnt, 1);
    // execute the kernel
   	step<<< blockLayout, threadLayout >>>(numParticles, positions, velocities, embedded,dt);
}
